#include "hip/hip_runtime.h"
/*
 * gpuMerge.cu
 *
 *  Created on: Dec 16, 2018
 *      Author: Orai Dezso Gergely
 */

#include "gpuMerge.cuh"
#include <iostream>
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


/**
 * CUDA kernel that sorts a float array
 */
__global__ void gpuMergeSortKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
void gpuMergeSort(float *data, unsigned size)
{
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	gpuMergeSortKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(data, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
}


/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
