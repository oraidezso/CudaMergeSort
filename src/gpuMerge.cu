#include "hip/hip_runtime.h"
/*
 * gpuMerge.cu
 *
 *  Created on: Dec 16, 2018
 *      Author: Orai Dezso Gergely
 */

#include "gpuMerge.cuh"
#include <iostream>
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
#define MIN_RUNTIME_VERSION 1000
#define MIN_COMPUTE_VERSION 0x10
int MaxThread = 512;

/**
 * CUDA kernel that sorts a float array
 */
__global__ void gpuMergeSortKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
void gpuMergeSort(float *data, unsigned size)
{
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = MaxThread;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	gpuMergeSortKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(data, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
}


/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}



bool findCudaDevice(){
	int deviceCount, bestDev=-1;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&deviceCount));
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        int runtimeVersion = 0;
        hipRuntimeGetVersion(&runtimeVersion);
        if (runtimeVersion >= MIN_RUNTIME_VERSION && ((deviceProp.major<<4) + deviceProp.minor) >= MIN_COMPUTE_VERSION)
        {
            if (bestDev == -1)
            {
                bestDev = dev;
                MaxThread = deviceProp.maxThreadsPerBlock;
            }
        }
    }
    return bestDev != -1;
}


